#include "hip/hip_runtime.h"
#include <cudatbx/math/special_functions/spherical_bessel_jn.cuh>

// large values may result in too few registers per thread
const int threads_per_block = 256;

namespace cudatbx {
namespace math {
namespace special_functions {

  /* ==========================================================================
     Implementation of the spherical Bessel function based on the explicit
     formula for integer order, n
                                  n/2
                                  ___
                         1        \       k  a_2k(n + 0.5)
       j_n(z) = sin(z - --- n pi) /   (-1)  --------------- +
                         2        ---         z^(2k + 1)
                                  k=0

                                (n-1)/2
                                  ___
                         1        \       k  a_2k+1 (n + 0.5)
                cos(z - --- n pi) /   (-1)  ------------------
                         2        ---         z^(2k + 2)
                                  k=0

     where
                           (n + k)!
       a_k(n + 0.5) = ----------------- , k = 0, 1, ... , n
                       2^k k! (n - k)!

                    = 0                 , k = n + 1, n + 2, ...

     In the limit that z -> 0, the limiting form,

                 n
       j_n(z) = z  / (2n + 1)!!

     is used because of the denominator in the summations (z^(some power)).

     The transition between the limiting form and the explicit formula was
     determined empirically up to order 50.  The largest errors occur around
     the transition point, but should be lower than 1e-4 for high orders
     (lower errors for low orders).

     Notation is taken from the NIST Digital Library of Mathematical Functions
     (http://dlmf.nist.gov/), sections 10.49 (Equations 10.49.1 and 10.49.2)
     and 10.52 (Equation 10.52.1).

     The output is formatted into chunks of z, that is, all j0(z) are first,
     followed by all j1(z), then j2(z), etc.
     --------------------------------------------------------------------------
  */

  __global__ void spherical_bessel_jn_kernel
  (const int order, const double* z, const int n_z, double* j) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n_z) {
      double z_i = z[i];
      for (int n=0; n<order+1; n++) {
        j[n*n_z + i] = spherical_bessel_jn<int,double>(n,z_i);
      }
    }
  }

  __device__ double2 spherical_bessel_j0_j1(double z) {
    double sin_z, cos_z;
    sincos(z,&sin_z,&cos_z);
    z = 1.0/z;
    sin_z = sin_z*z;
    return make_double2(sin_z,sin_z*z - cos_z*z);
  }

  scitbx::af::shared<double> cuda_spherical_bessel_jn
  (const int& order, const scitbx::af::const_ref<double>& z) {

    // allocate and initialize arrays
    int n_order = order + 1;
    int j_size = n_order * z.size();
    double * h_z, * h_j;
    double * d_z, * d_j;
    h_z = (double*)&z[0];
    h_j = new double[j_size];
    cudaSafeCall( hipMalloc( (void**)&d_z, z.size() * sizeof(double) ) );
    cudaSafeCall( hipMalloc( (void**)&d_j, j_size * sizeof(double) ) );
    cudaSafeCall( hipMemcpy( d_z, h_z, z.size() * sizeof(double),
                              hipMemcpyHostToDevice ) );

    // run kernel
    int blocks_per_grid = (z.size() + threads_per_block - 1)/threads_per_block;
    spherical_bessel_jn_kernel<<<blocks_per_grid,threads_per_block>>>
      (order,d_z,z.size(),d_j);

    // copy result from GPU
    cudaSafeCall( hipMemcpy( h_j, d_j, j_size * sizeof(double),
                              hipMemcpyDeviceToHost ) );
    scitbx::af::shared<double> jn((double*)&h_j[0], (double*)&h_j[0] + j_size);

    // clean up
    delete h_j;
    cudaSafeCall( hipFree( d_z ) );
    cudaSafeCall( hipFree( d_j ) );

    return jn;
  }

}
}
}
