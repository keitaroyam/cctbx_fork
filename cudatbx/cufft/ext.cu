
#include <boost/python/module.hpp>
#include <boost/python/def.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace cudatbx { namespace cufft {
  void clean_up () {
    hipDeviceReset();
  }

  void wrap_cufft_single_precision();
  void wrap_cufft_double_precision();
  void wrap_util () {
    using namespace boost::python;
    def("clean_up", clean_up);
  }

namespace {
  void init_module() {
    wrap_cufft_single_precision();
    wrap_cufft_double_precision();
    wrap_util();
  }
}
}}

BOOST_PYTHON_MODULE(cudatbx_cufft_ext)
{
  cudatbx::cufft::init_module();
}
