#include <cudatbx/cuda_base.h>

namespace cudatbx {

  int number_of_gpus() {
    int device_count = 0;
    cudaSafeCall( hipGetDeviceCount(&device_count) );
    return device_count;
  }

  void reset_gpu(const int& gpu_id) {
    cudaSafeCall( hipSetDevice(gpu_id) );
    cudaSafeCall( hipDeviceReset() );
  }

}
