#include <cudatbx/cuda_base.h>

namespace cudatbx {

  int number_of_gpus() {
    int device_count = 0;
    cudaSafeCall( hipGetDeviceCount(&device_count) );
    return device_count;
  }

  void reset_gpu(const int& gpu_id) {
    int device_count = number_of_gpus();
    if (gpu_id < device_count) {
      cudaSafeCall( hipSetDevice(gpu_id) );
      cudaSafeCall( hipDeviceReset() );
    }
    else {
      std::cerr << "WARNING: Device " << gpu_id << " does not exist.\n";
    }
  }

}
