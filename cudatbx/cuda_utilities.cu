#include <cudatbx/cuda_utilities.h>
#include <cudatbx/cuda_base.cuh>

namespace cudatbx {

  int number_of_gpus() {
    int device_count = 0;
    cudaSafeCall( hipGetDeviceCount(&device_count) );
    return device_count;
  }

  void reset_gpu(const int& gpu_id) {
    cudaSafeCall( hipSetDevice(gpu_id) );
    cudaSafeCall( hipDeviceReset() );
  }

}
