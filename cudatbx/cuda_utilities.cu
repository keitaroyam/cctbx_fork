#include "hip/hip_runtime.h"
#include <cudatbx/cuda_utilities.cuh>

namespace cudatbx {

  int number_of_gpus() {
    int device_count = 0;
    cudaSafeCall( hipGetDeviceCount(&device_count) );
    return device_count;
  }

  void reset_gpu(const int& gpu_id) {
    cudaSafeCall( hipSetDevice(gpu_id) );
    cudaSafeCall( hipDeviceReset() );
  }

  int calculate_padded_size(const int& size, const int& padding) {
    int padded_size = int(std::floor(size/padding + 1.0)) * padding;
    return padded_size;
  }

  int calculate_blocks_per_grid(const int& size, const int& threads_per_block) {
    int blocks_per_grid = (size + threads_per_block - 1)/threads_per_block;
    return blocks_per_grid;
  }

  /* ==========================================================================
     Basic timer for CUDA using events, one stream only, no checks

     Usage:

       cuda_timer t;
       t.start();

       < run CUDA stuff >

       t.stop();
       float elapsed_time = t.get_elapsed_time();
       std::cout << elapsed_time << "\n";
  */
  cudatbx::cuda_timer::cuda_timer() {
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
  }

  cudatbx::cuda_timer::~cuda_timer() {
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
  }

  void cudatbx::cuda_timer::start() {
    hipEventRecord(start_event);
  }

  void cudatbx::cuda_timer::stop() {
    hipEventRecord(stop_event);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);
  }

  float cudatbx::cuda_timer::get_elapsed_time() {
    return elapsed_time;
  }

  // ==========================================================================

}
