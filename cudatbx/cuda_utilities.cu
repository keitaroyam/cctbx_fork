#include <cudatbx/cuda_utilities.cuh>

namespace cudatbx {

  int number_of_gpus() {
    int device_count = 0;
    cudaSafeCall( hipGetDeviceCount(&device_count) );
    return device_count;
  }

  void reset_gpu(const int& gpu_id) {
    cudaSafeCall( hipSetDevice(gpu_id) );
    cudaSafeCall( hipDeviceReset() );
  }

  /* ==========================================================================
     Basic timer for CUDA using events, one stream only, no checks

     Usage:

       cuda_timer t;
       t.start();

       < run CUDA stuff >

       t.stop();
       float elapsed_time = t.get_elapsed_time();
       std::cout << elapsed_time << "\n";
  */
  cudatbx::cuda_timer::cuda_timer() {
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
  }

  cudatbx::cuda_timer::~cuda_timer() {
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
  }

  void cudatbx::cuda_timer::start() {
    hipEventRecord(start_event);
  }

  void cudatbx::cuda_timer::stop() {
    hipEventRecord(stop_event);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);
  }

  float cudatbx::cuda_timer::get_elapsed_time() {
    return elapsed_time;
  }

  // ==========================================================================

}
