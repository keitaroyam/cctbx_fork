#include "hip/hip_runtime.h"
#include <cudatbx/scattering/direct_summation.cuh>

namespace cudatbx {
namespace scattering {

  /* ==========================================================================
   */
  cudatbx::scattering::direct_summation::direct_summation() {
    // set host and device pointers to NULL
    h_xyz = NULL;
    h_solvent = NULL;
    h_h = NULL;
    h_rt = NULL;
    h_weights = NULL;
    h_scattering_type = NULL;
    h_a = NULL;
    h_b = NULL;
    h_c = NULL;

    d_xyz = NULL;
    d_solvent = NULL;
    d_h = NULL;
    d_rt = NULL;
    d_weights = NULL;
    d_scattering_type = NULL;

    amplitudes_allocated = false;
    h_real = NULL;
    h_imag = NULL;
    d_real = NULL;
    d_imag = NULL;

    workspace_allocated = false;
    d_workspace = NULL;
  }

  cudatbx::scattering::direct_summation::~direct_summation() {
    clear_arrays();
    clear_amplitudes();
    clear_workspace();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::set_xyz
  (const scitbx::af::const_ref<scitbx::vec3<double> >& xyz) {
    // allocate memory if necessary
    if (n_xyz != xyz.size()) {
      clear_xyz();
      n_xyz = xyz.size();
      padded_n_xyz = cudatbx::calculate_padded_size(n_xyz,padding);
      size_xyz = 3 * padded_n_xyz;
      h_xyz = new fType[size_xyz];
      cudaSafeCall( hipMalloc((void**)&d_xyz,size_xyz*sizeof(fType)) );
    }

    // convert values
    for (int i=0; i<n_xyz; i++) {
      for (int j=0; j<3; j++) {
        h_xyz[j*padded_n_xyz + i] = fType(xyz[i][j]);
      }
    }

    // transfer to GPU
    cudaSafeCall( hipMemcpy(d_xyz, h_xyz, size_xyz*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_xyz() {
    delete[] h_xyz;
    cudaSafeCall( hipFree(d_xyz) );
    h_xyz = NULL;
    d_xyz = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::set_solvent_weights
  (const scitbx::af::const_ref<double>& solvent_weights) {
    // allocate memory if necessary
    SCITBX_ASSERT (n_xyz == solvent_weights.size());
    if (n_solvent != solvent_weights.size()) {
      clear_solvent_weights();
      n_solvent = solvent_weights.size();
      h_solvent = new fType[padded_n_xyz];
      cudaSafeCall( hipMalloc((void**)&d_solvent,padded_n_xyz*sizeof(fType)) );
    }

    // convert values
    for (int i=0; i<n_xyz; i++) {
      h_solvent[i] = fType(solvent_weights[i]);
    }

    // transfer to GPU
    cudaSafeCall( hipMemcpy(d_solvent, h_solvent,
                             padded_n_xyz*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_solvent_weights() {
    delete[] h_solvent;
    cudaSafeCall( hipFree(d_solvent) );
    h_solvent = NULL;
    d_solvent = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::set_hkl
  (const scitbx::af::const_ref<scitbx::vec3<double> >& h) {
    // allocate memory if necessary
    if (n_h != h.size()) {
      clear_hkl();
      n_h = h.size();
      padded_n_h = cudatbx::calculate_padded_size(n_h,padding);
      size_h = 3 * padded_n_h;
      h_h = new fType[size_h];
      cudaSafeCall( hipMalloc((void**)&d_h,size_h*sizeof(fType)) );
    }

    // convert values
    for (int i=0; i<n_h; i++) {
      for (int j=0; j<3; j++) {
        h_h[j*padded_n_h + i] = fType(h[i][j]);
      }
    }

    // transfer to GPU
    cudaSafeCall( hipMemcpy(d_h, h_h, size_h*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_hkl() {
    delete[] h_h;
    cudaSafeCall( hipFree(d_h) );
    h_h = NULL;
    d_h = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::set_q
  (const scitbx::af::const_ref<double>& q) {
    // q data, use h variables
    if (n_h != q.size()) {
      clear_q();
      n_h = q.size();
      padded_n_h = cudatbx::calculate_padded_size(n_h,padding);
      size_h = padded_n_h;
      h_h = new fType[size_h];
      cudaSafeCall( hipMalloc((void**)&d_h,size_h*sizeof(fType)) );
    }

    // convert values
    for (int i=0; i<n_h; i++) {
      h_h[i] = fType(q[i]);
    }

    // transfer to GPU
    cudaSafeCall( hipMemcpy(d_h, h_h, size_h*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_q() {
    clear_hkl();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::set_lattice
  (const scitbx::af::const_ref<double>& lattice_weights,
   const scitbx::af::const_ref<double>& lattice) {
    // lattice points, use rotation/translation
    if (n_rt != lattice_weights.size()) {
      clear_lattice();
      n_rt = lattice_weights.size();
      size_rt = cudatbx::calculate_padded_size(n_rt,padding);
      h_weights = new fType[size_rt];
      h_rt = new fType[3*size_rt];
      cudaSafeCall( hipMalloc((void**)&d_weights,size_rt*sizeof(fType)) );
      cudaSafeCall( hipMalloc((void**)&d_rt,3*size_rt*sizeof(fType)) );
    }

    // convert values
    for (int i=0; i<n_rt; i++) {
      h_weights[i] = fType(lattice_weights[i]/n_rt);
      for (int j=0; j<3; j++) {
        h_rt[j*size_rt + i] = fType(lattice[j*n_rt + i]);
      }
    }

    // transfer to GPU
    cudaSafeCall( hipMemcpy(d_weights, h_weights, size_rt*sizeof(fType),
                             hipMemcpyHostToDevice) );
    cudaSafeCall( hipMemcpy(d_rt, h_rt, 3*size_rt*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_weights() {
    delete[] h_weights;
    cudaSafeCall( hipFree(d_weights) );
    h_weights = NULL;
    d_weights = NULL;
  }

  void cudatbx::scattering::direct_summation::clear_lattice() {
    clear_weights();
    clear_rotations_translations();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::set_rotations_translations
  (const scitbx::af::const_ref<double>& rotations,
   const scitbx::af::const_ref<scitbx::vec3<double> >& translations) {
    // each rotation/translation pair is combined and padded to take up
    // 64 bytes so that a coalesced read will read two pairs
    if (n_rt != translations.size()) {
      clear_rotations_translations();
      n_rt = translations.size();
      size_rt = padded_size * n_rt;
      h_rt = new fType[size_rt];
      cudaSafeCall( hipMalloc((void**)&d_rt,size_rt*sizeof(fType)) );
    }

    // convert values
    for (int i=0; i<n_rt; i++) {
      for (int j=0; j<9; j++) {
        h_rt[padded_size*i + j] = fType(rotations[9*i + j]);
      }
      for (int j=0; j<3; j++) {
        h_rt[padded_size*i + j + 9] = fType(translations[i][j]);
      }
    }

    // transfer to GPU
    cudaSafeCall( hipMemcpy(d_rt, h_rt, size_rt*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_rotations_translations() {
    delete[] h_rt;
    cudaSafeCall( hipFree(d_rt) );
    h_rt = NULL;
    d_rt = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::set_scattering_types
  (const scitbx::af::const_ref<std::string>& scatterers,
   const cctbx::xray::scattering_type_registry& registry) {
    // allocate memory if necessary
    SCITBX_ASSERT (n_xyz == scatterers.size());
    if (n_scatterers != scatterers.size()) {
      clear_scattering_types();
      n_scatterers = scatterers.size();
      h_scattering_type = new int[padded_n_xyz];
      cudaSafeCall( hipMalloc((void**)&d_scattering_type,
                               padded_n_xyz*sizeof(int)) );
    }

    // convert values
    for (int i=0; i<n_xyz; i++) {
      h_scattering_type[i] = registry.unique_index(scatterers[i]);
    }

    // transfer to GPU
    cudaSafeCall( hipMemcpy(d_scattering_type,h_scattering_type,
                             padded_n_xyz*sizeof(int),hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_scattering_types() {
    delete[] h_scattering_type;
    cudaSafeCall( hipFree(d_scattering_type) );
    h_scattering_type = NULL;
    d_scattering_type = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::set_scattering_type_registry
  (const cctbx::xray::scattering_type_registry& registry,
   const bool& complex_form_factor) {
    // convert form factors
    // add ordinary oxygen form factor at end for boundary layer solvent
    scitbx::af::shared<boost::optional
                       <cctbx::eltbx::xray_scattering::gaussian> >
      unique_gaussians = registry.unique_gaussians;
    n_types = unique_gaussians.size() + 1;
    n_terms = unique_gaussians[0].get().n_terms();
    f_size = n_types * n_terms;
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    h_a = new fType[f_size];
    h_b = new fType[f_size];
    h_c = new fType[n_types];
    for (int i=0; i<f_size; i++) {
      h_a[i] = fType(0.0);
      h_b[i] = fType(0.0);
    }
    for (int i=0; i<n_types-1; i++) {
      for (int j=0; j<n_terms; j++) {
        h_a[i*n_terms + j] = fType(unique_gaussians[i].get().array_of_a()[j]);
        h_b[i*n_terms + j] = fType(unique_gaussians[i].get().array_of_b()[j]);
      }
      if (unique_gaussians[i].get().use_c()) {
        h_c[i] = fType(unique_gaussians[i].get().c());
      }
      else {
        h_c[i] = fType(0.0);
      }
    }

    // add form factor for boundary layer solvent
    cctbx::eltbx::xray_scattering::gaussian hoh =
      cctbx::eltbx::xray_scattering::wk1995("O",true).fetch();
    for (int i=0; i<hoh.array_of_a().size(); i++){
      h_a[(n_types-1)*n_terms + i] = fType(hoh.array_of_a()[i]);
      h_b[(n_types-1)*n_terms + i] = fType(hoh.array_of_b()[i]);
    }
    if (hoh.use_c()) {
      h_c[n_types-1] = fType(hoh.c());
    }
    else {
      h_c[n_types-1] = fType(0.0);
    }

    // transfer to GPU
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_a), h_a, f_size*sizeof(fType)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_b), h_b, f_size*sizeof(fType)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_c), h_c, n_types*sizeof(fType)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_n_types), &n_types, sizeof(int)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_n_terms), &n_terms, sizeof(int)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_complex_form_factor),
                                     &complex_form_factor, sizeof(bool)) );
  }

  void cudatbx::scattering::direct_summation::clear_scattering_type_registry() {
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    h_a = NULL;
    h_b = NULL;
    h_c = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::clear_arrays() {
    // clear pointers and set all pointers to NULL
    clear_xyz();
    clear_solvent_weights();
    clear_hkl();
    clear_rotations_translations();
    clear_weights();
    clear_scattering_types();
    clear_scattering_type_registry();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::allocate_amplitudes() {
    if (amplitudes_allocated) {
      clear_amplitudes();
    }
    h_real = new fType[n_h];
    h_imag = new fType[n_h];
    cudaSafeCall( hipMalloc((void**)&d_real,n_h*sizeof(fType)) );
    cudaSafeCall( hipMalloc((void**)&d_imag,n_h*sizeof(fType)) );
    amplitudes_allocated = true;
  }

  void cudatbx::scattering::direct_summation::reset_amplitudes() {
    fType zero = fType(0.0);
    for (int i=0; i<n_h; i++) {
      h_real[i] = zero;
      h_imag[i] = zero;
    }
    cudaSafeCall( hipMemcpy(d_real,h_real,n_h*sizeof(fType),
                             hipMemcpyHostToDevice) );
    cudaSafeCall( hipMemcpy(d_imag,h_imag,n_h*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_amplitudes() {
    delete[] h_real;
    delete[] h_imag;
    cudaSafeCall( hipFree(d_real) );
    cudaSafeCall( hipFree(d_imag) );
    h_real = NULL;
    h_imag = NULL;
    d_real = NULL;
    d_imag = NULL;
    amplitudes_allocated = false;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::allocate_workspace
  (const int& length) {
    if (workspace_allocated) {
      clear_workspace();
    }
    cudaSafeCall( hipMalloc((void**)&d_workspace,length*sizeof(fType)) );
    workspace_allocated = true;
  }

  void cudatbx::scattering::direct_summation::clear_workspace() {
    cudaSafeCall( hipFree(d_workspace) );
    d_workspace = NULL;
    workspace_allocated = false;
  }

  /* --------------------------------------------------------------------------
     reorganizes data and calls cuda
     padded to multiple of 128 bytes, (32 * sizeof(float or int))
  */
  void cudatbx::scattering::direct_summation::run_kernel() {
    int blocks_per_grid = cudatbx::calculate_blocks_per_grid(n_h,threads_per_block);
    structure_factor_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (d_scattering_type, d_xyz, d_solvent, n_xyz, padded_n_xyz,
       d_h, n_h, padded_n_h,
       d_rt, n_rt,
       d_real, d_imag);
  }

  void cudatbx::scattering::direct_summation::add
  (const scitbx::af::const_ref<std::string>& scatterers,
   const scitbx::af::const_ref<scitbx::vec3<double> >& xyz,
   const scitbx::af::const_ref<double>& solvent_weights,
   const scitbx::af::const_ref<scitbx::vec3<double> >& h,
   const scitbx::af::const_ref<double>& rotations,
   const scitbx::af::const_ref<scitbx::vec3<double> >& translations,
   const cctbx::xray::scattering_type_registry& registry,
   const bool& complex_form_factor) {

    // reorganize input data, allocates arrays, transfer to GPU, order matters
    set_xyz(xyz);
    set_solvent_weights(solvent_weights);
    set_hkl(h);
    set_rotations_translations(rotations,translations);
    set_scattering_types(scatterers,registry);
    set_scattering_type_registry(registry,complex_form_factor);

    // allocate arrays for results if necessary
    if (!amplitudes_allocated) {
      allocate_amplitudes();
      reset_amplitudes();
    }

    // run calculation
    run_kernel();

    // deallocate arrays
    clear_arrays();
  }

  /* --------------------------------------------------------------------------
     reorganizes data and calls cuda
     padded to multiple of 128 bytes, (32 * sizeof(float or int))

     "Rapid and accurate calculation of small-angle scattering profiles using
      the golden ratio"
     Watson, MC, Curtis, JE. J. Appl. Cryst. (2013). 46, 1171-1177

     solvent variables are used for weights and code is not optimal
     possibly subclass or split everything into functions
  */
  void cudatbx::scattering::direct_summation::prepare_saxs
  (const scitbx::af::const_ref<std::string>& scatterers,
   const scitbx::af::const_ref<scitbx::vec3<double> >& xyz,
   const scitbx::af::const_ref<double>& solvent_weights,
   const scitbx::af::const_ref<double>& q,
   const scitbx::af::const_ref<double>& lattice_weights,
   const scitbx::af::const_ref<double>& lattice,
   const cctbx::xray::scattering_type_registry& registry,
   const bool& complex_form_factor) {

    // reorganize input data, allocates arrays, transfer to GPU, order matters
    set_xyz(xyz);
    set_solvent_weights(solvent_weights);
    set_q(q);
    set_lattice(lattice_weights,lattice);
    set_scattering_types(scatterers,registry);
    set_scattering_type_registry(registry,complex_form_factor);

    // allocate arrays for results if necessary
    if (!amplitudes_allocated) {
      allocate_amplitudes();
    }
  }

  void cudatbx::scattering::direct_summation::run_saxs_kernel() {
    // allocate working space if necessary
    if (!workspace_allocated) {
      workspace_size = int(std::floor(n_h*n_rt/padding + 1.0)) * padding;
      allocate_workspace(3*workspace_size);
    }

    int blocks_per_grid = cudatbx::calculate_blocks_per_grid
      (n_rt,threads_per_block);
    expand_q_lattice_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (d_h, n_h,
       d_rt, n_rt, size_rt,
       d_workspace, workspace_size);
    blocks_per_grid = cudatbx::calculate_blocks_per_grid
      (n_h*n_rt,threads_per_block);
    saxs_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (d_scattering_type, d_xyz, d_solvent, n_xyz, padded_n_xyz,
       n_h, n_rt,
       d_workspace, workspace_size);
  }

  void cudatbx::scattering::direct_summation::run_solvent_saxs_kernel() {
    // allocate working space if necessary
    if (!workspace_allocated) {
      workspace_size = int(std::floor(n_h*n_rt/padding + 1.0)) * padding;
      allocate_workspace(7*workspace_size);
    }

    int blocks_per_grid = cudatbx::calculate_blocks_per_grid
      (n_rt,threads_per_block);
    expand_q_lattice_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (d_h, n_h,
       d_rt, n_rt, size_rt,
       d_workspace, workspace_size);
    blocks_per_grid = cudatbx::calculate_blocks_per_grid
      (n_h*n_rt,threads_per_block);
    solvent_saxs_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (d_scattering_type, d_xyz, d_solvent, n_xyz, padded_n_xyz,
       n_h, n_rt,
       d_workspace, workspace_size);
  }

  void cudatbx::scattering::direct_summation::run_collect_solvent_saxs_kernel
  (const double& c1, const double& c2) {
    assert(workspace_allocated);
    int blocks_per_grid = cudatbx::calculate_blocks_per_grid
      (n_h*n_rt,threads_per_block);
    collect_solvent_saxs_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (n_h, n_rt,fType(c1),fType(c2),d_workspace, workspace_size);
  }

  void cudatbx::scattering::direct_summation::sum_over_lattice() {
    int blocks_per_grid = cudatbx::calculate_blocks_per_grid
      (n_rt,threads_per_block);
    for (int i=0; i<n_h; i++) {
      cudatbx::math::weighted_sum_kernel<fType>
        <<<blocks_per_grid,threads_per_block,threads_per_block*sizeof(fType)>>>
        (&d_workspace[i*n_rt],d_weights,n_rt,&d_real[i]);
    }
  }

  /* --------------------------------------------------------------------------
     return total sum
  */
  scitbx::af::shared<std::complex<double> >
  cudatbx::scattering::direct_summation::get_sum() {
    scitbx::af::shared<std::complex<double> > sf(n_h);
    assert(amplitudes_allocated);
    cudaSafeCall( hipMemcpy(h_real,d_real,n_h*sizeof(fType),
                             hipMemcpyDeviceToHost) );
    cudaSafeCall( hipMemcpy(h_imag,d_imag,n_h*sizeof(fType),
                             hipMemcpyDeviceToHost) );
    for (int i=0; i<n_h; i++) {
      sf[i] = std::complex<double>(double(h_real[i]),double(h_imag[i]));
    }

    return sf;
  }

  /* ==========================================================================
   */

}
}
