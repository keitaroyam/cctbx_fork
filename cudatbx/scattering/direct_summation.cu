#include "hip/hip_runtime.h"
#include <cudatbx/scattering/direct_summation.cuh>

namespace cudatbx {
namespace scattering {

  /* ==========================================================================
   */
  cudatbx::scattering::direct_summation::direct_summation() {
    // set host and device pointers to NULL
    h_xyz = NULL;
    h_solvent = NULL;
    h_h = NULL;
    h_rt = NULL;
    h_weights = NULL;
    h_scattering_type = NULL;
    h_a = NULL;
    h_b = NULL;
    h_c = NULL;

    d_xyz = NULL;
    d_solvent = NULL;
    d_h = NULL;
    d_rt = NULL;
    d_weights = NULL;
    d_scattering_type = NULL;

    amplitudes_allocated = false;
    h_real = NULL;
    h_imag = NULL;
    d_real = NULL;
    d_imag = NULL;

    workspace_allocated = false;
    d_workspace = NULL;
  }

  cudatbx::scattering::direct_summation::~direct_summation() {
    clear_arrays();
    clear_amplitudes();
    clear_workspace();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::reorganize_xyz
  (const scitbx::af::const_ref<scitbx::vec3<double> >& xyz) {
    n_xyz = xyz.size();
    padded_n_xyz = int(std::floor(n_xyz/padding + 1.0)) * padding;
    size_xyz = 3 * padded_n_xyz;
    delete[] h_xyz;
    h_xyz = new fType[size_xyz];
    for (int i=0; i<n_xyz; i++) {
      for (int j=0; j<3; j++) {
        h_xyz[j*padded_n_xyz + i] = fType(xyz[i][j]);
      }
    }
  }

  void cudatbx::scattering::direct_summation::transfer_xyz() {
    cudaSafeCall( hipMalloc((void**)&d_xyz,size_xyz*sizeof(fType)) );
    cudaSafeCall( hipMemcpy(d_xyz, h_xyz, size_xyz*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_xyz() {
    delete[] h_xyz;
    cudaSafeCall( hipFree(d_xyz) );
    h_xyz = NULL;
    d_xyz = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::copy_solvent_weights
  (const scitbx::af::const_ref<double>& solvent_weights) {
    delete[] h_solvent;
    h_solvent = new fType[padded_n_xyz];
    for (int i=0; i<n_xyz; i++) {
      h_solvent[i] = fType(solvent_weights[i]);
    }
  }

  void cudatbx::scattering::direct_summation::transfer_solvent_weights() {
    cudaSafeCall( hipMalloc((void**)&d_solvent,padded_n_xyz*sizeof(fType)) );
    cudaSafeCall( hipMemcpy(d_solvent, h_solvent,
                             padded_n_xyz*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_solvent_weights() {
    delete[] h_solvent;
    cudaSafeCall( hipFree(d_solvent) );
    h_solvent = NULL;
    d_solvent = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::reorganize_coordinates
  (const scitbx::af::const_ref<scitbx::vec3<double> >& xyz,
   const scitbx::af::const_ref<double>& solvent_weights) {
    reorganize_xyz(xyz);
    transfer_xyz();

    SCITBX_ASSERT (solvent_weights.size() == n_xyz);
    copy_solvent_weights(solvent_weights);
    transfer_solvent_weights();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::reorganize_hkl
  (const scitbx::af::const_ref<scitbx::vec3<double> >& h) {
    n_h = h.size();
    padded_n_h = int(std::floor(n_h/padding + 1.0)) * padding;
    size_h = 3 * padded_n_h;
    delete[] h_h;
    h_h = new fType[size_h];
    for (int i=0; i<n_h; i++) {
      for (int j=0; j<3; j++) {
        h_h[j*padded_n_h + i] = fType(h[i][j]);
      }
    }    
  }

  void cudatbx::scattering::direct_summation::transfer_hkl() {
    cudaSafeCall( hipMalloc((void**)&d_h,size_h*sizeof(fType)) );
    cudaSafeCall( hipMemcpy(d_h, h_h, size_h*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_hkl() {
    delete[] h_h;
    cudaSafeCall( hipFree(d_h) );
    h_h = NULL;
    d_h = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::reorganize_h
  (const scitbx::af::const_ref<scitbx::vec3<double> >& h) {
    reorganize_hkl(h);
    transfer_hkl();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::copy_q
  (const scitbx::af::const_ref<double>& q) {
    // q data, use h variables
    n_h = q.size();
    padded_n_h = int(std::floor(n_h/padding + 1.0)) * padding;
    size_h = padded_n_h;
    delete[] h_h;
    h_h = new fType[size_h];
    for (int i=0; i<n_h; i++) {
      h_h[i] = fType(q[i]);
    }
  }

  void cudatbx::scattering::direct_summation::transfer_q() {
    cudaSafeCall( hipMalloc((void**)&d_h,size_h*sizeof(fType)) );
    cudaSafeCall( hipMemcpy(d_h, h_h, size_h*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_q() {
    clear_hkl();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::copy_lattice
  (const scitbx::af::const_ref<double>& lattice_weights,
   const scitbx::af::const_ref<double>& lattice) {
    // lattice points, use rotation/translation
    n_rt = lattice_weights.size();
    size_rt = int(std::floor(n_rt/padding + 1.0)) * padding;
    delete[] h_weights;
    delete[] h_rt;
    h_weights = new fType[size_rt];
    h_rt = new fType[3*size_rt];
    for (int i=0; i<n_rt; i++) {
      h_weights[i] = fType(lattice_weights[i]);
      for (int j=0; j<3; j++) {
        h_rt[j*size_rt + i] = fType(lattice[j*n_rt + i]);
      }
    }
  }

  void cudatbx::scattering::direct_summation::transfer_lattice() {
    cudaSafeCall( hipMalloc((void**)&d_weights,size_rt*sizeof(fType)) );
    cudaSafeCall( hipMemcpy(d_weights, h_weights, size_rt*sizeof(fType),
                             hipMemcpyHostToDevice) );
    cudaSafeCall( hipMalloc((void**)&d_rt,3*size_rt*sizeof(fType)) );
    cudaSafeCall( hipMemcpy(d_rt, h_rt, 3*size_rt*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_weights() {
    delete[] h_weights;
    cudaSafeCall( hipFree(d_weights) );
    h_weights = NULL;
    d_weights = NULL;
  }

  void cudatbx::scattering::direct_summation::clear_lattice() {
    clear_weights();
    clear_rotations_translations();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::reorganize_q
  (const scitbx::af::const_ref<double>& q,
   const scitbx::af::const_ref<double>& lattice_weights,
   const scitbx::af::const_ref<double>& lattice) {
    copy_q(q);
    transfer_q();

    copy_lattice(lattice_weights,lattice);
    transfer_lattice();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::reorganize_rotations_translations
  (const scitbx::af::const_ref<double>& rotations,
   const scitbx::af::const_ref<scitbx::vec3<double> >& translations) {
    // each rotation/translation pair is combined and padded to take up
    // 64 bytes so that a coalesced read will read two pairs
    n_rt = translations.size();
    size_rt = padded_size * n_rt;
    delete[] h_rt;
    h_rt = new fType[size_rt];
    for (int i=0; i<n_rt; i++) {
      for (int j=0; j<9; j++) {
        h_rt[padded_size*i + j] = fType(rotations[9*i + j]);
      }
      for (int j=0; j<3; j++) {
        h_rt[padded_size*i + j + 9] = fType(translations[i][j]);
      }
    }
  }

  void cudatbx::scattering::direct_summation::transfer_rotations_translations() {
    cudaSafeCall( hipMalloc((void**)&d_rt,size_rt*sizeof(fType)) );
    cudaSafeCall( hipMemcpy(d_rt, h_rt, size_rt*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_rotations_translations() {
    delete[] h_rt;
    cudaSafeCall( hipFree(d_rt) );
    h_rt = NULL;
    d_rt = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::reorganize_rt
  (const scitbx::af::const_ref<double>& rotations,
   const scitbx::af::const_ref<scitbx::vec3<double> >& translations) {
    reorganize_rotations_translations(rotations,translations);
    transfer_rotations_translations();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::convert_scattering_types
  (const scitbx::af::const_ref<std::string>& scatterers,
   const cctbx::xray::scattering_type_registry& registry) {
    // convert scattering types
    delete[] h_scattering_type;
    h_scattering_type = new int[padded_n_xyz];
    for (int i=0; i<n_xyz; i++) {
      h_scattering_type[i] = registry.unique_index(scatterers[i]);
    }
  }

  void cudatbx::scattering::direct_summation::transfer_scattering_types() {
    cudaSafeCall( hipMalloc((void**)&d_scattering_type,padded_n_xyz*sizeof(int)) );
    cudaSafeCall( hipMemcpy(d_scattering_type,h_scattering_type,
                             padded_n_xyz*sizeof(int),hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_scattering_types() {
    delete[] h_scattering_type;
    cudaSafeCall( hipFree(d_scattering_type) );
    h_scattering_type = NULL;
    d_scattering_type = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::convert_scattering_type_registry
  (const cctbx::xray::scattering_type_registry& registry) {
    // convert form factors
    // add ordinary oxygen form factor at end for boundary layer solvent
    scitbx::af::shared<boost::optional
                       <cctbx::eltbx::xray_scattering::gaussian> >
      unique_gaussians = registry.unique_gaussians;
    n_types = unique_gaussians.size() + 1;
    n_terms = unique_gaussians[0].get().n_terms();
    f_size = n_types * n_terms;
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    h_a = new fType[f_size];
    h_b = new fType[f_size];
    h_c = new fType[n_types];
    for (int i=0; i<f_size; i++) {
      h_a[i] = fType(0.0);
      h_b[i] = fType(0.0);
    }
    for (int i=0; i<n_types-1; i++) {
      for (int j=0; j<n_terms; j++) {
        h_a[i*n_terms + j] = fType(unique_gaussians[i].get().array_of_a()[j]);
        h_b[i*n_terms + j] = fType(unique_gaussians[i].get().array_of_b()[j]);
      }
      if (unique_gaussians[i].get().use_c()) {
        h_c[i] = fType(unique_gaussians[i].get().c());
      }
      else {
        h_c[i] = fType(0.0);
      }
    }

    // add form factor for boundary layer solvent
    cctbx::eltbx::xray_scattering::gaussian hoh =
      cctbx::eltbx::xray_scattering::wk1995("O",true).fetch();
    for (int i=0; i<hoh.array_of_a().size(); i++){
      h_a[(n_types-1)*n_terms + i] = fType(hoh.array_of_a()[i]);
      h_b[(n_types-1)*n_terms + i] = fType(hoh.array_of_b()[i]);
    }
    if (hoh.use_c()) {
      h_c[n_types-1] = fType(hoh.c());
    }
    else {
      h_c[n_types-1] = fType(0.0);
    }
  }

  void cudatbx::scattering::direct_summation::transfer_scattering_type_registry
  (const bool& complex_form_factor) {
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_a), h_a, f_size*sizeof(fType)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_b), h_b, f_size*sizeof(fType)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_c), h_c, n_types*sizeof(fType)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_n_types), &n_types, sizeof(int)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_n_terms), &n_terms, sizeof(int)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_complex_form_factor),
                                     &complex_form_factor, sizeof(bool)) );
  }

  void cudatbx::scattering::direct_summation::clear_scattering_type_registry() {
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    h_a = NULL;
    h_b = NULL;
    h_c = NULL;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::convert_scatterers
  (const scitbx::af::const_ref<std::string>& scatterers,
   const cctbx::xray::scattering_type_registry& registry,
   const bool& complex_form_factor) {
    convert_scattering_types(scatterers,registry);
    transfer_scattering_types();

    convert_scattering_type_registry(registry);
    transfer_scattering_type_registry(complex_form_factor);
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::clear_arrays() {
    // clear pointers and set all pointers to NULL
    clear_xyz();
    clear_solvent_weights();
    clear_hkl();
    clear_rotations_translations();
    clear_weights();
    clear_scattering_types();
    clear_scattering_type_registry();
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::allocate_amplitudes() {
    h_real = new fType[n_h];
    h_imag = new fType[n_h];
    cudaSafeCall( hipMalloc((void**)&d_real,n_h*sizeof(fType)) );
    cudaSafeCall( hipMalloc((void**)&d_imag,n_h*sizeof(fType)) );
    amplitudes_allocated = true;
  }

  void cudatbx::scattering::direct_summation::reset_amplitudes() {
    fType zero = fType(0.0);
    for (int i=0; i<n_h; i++) {
      h_real[i] = zero;
      h_imag[i] = zero;
    }
    cudaSafeCall( hipMemcpy(d_real,h_real,n_h*sizeof(fType),
                             hipMemcpyHostToDevice) );
    cudaSafeCall( hipMemcpy(d_imag,h_imag,n_h*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  void cudatbx::scattering::direct_summation::clear_amplitudes() {
    delete[] h_real;
    delete[] h_imag;
    cudaSafeCall( hipFree(d_real) );
    cudaSafeCall( hipFree(d_imag) );
    h_real = NULL;
    h_imag = NULL;
    d_real = NULL;
    d_imag = NULL;
    amplitudes_allocated = false;
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::allocate_workspace
  (const int& length) {
    cudaSafeCall( hipMalloc((void**)&d_workspace,length*sizeof(fType)) );
    workspace_allocated = true;
  }

  void cudatbx::scattering::direct_summation::clear_workspace() {
    cudaSafeCall( hipFree(d_workspace) );
    d_workspace = NULL;
    workspace_allocated = false;
  }

  /* --------------------------------------------------------------------------
     reorganizes data and calls cuda
     padded to multiple of 128 bytes, (32 * sizeof(float or int))
  */
  void cudatbx::scattering::direct_summation::run_kernel() {
    int blocks_per_grid = (n_h + threads_per_block - 1)/threads_per_block;
    structure_factor_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (d_scattering_type, d_xyz, d_solvent, n_xyz, padded_n_xyz,
       d_h, n_h, padded_n_h,
       d_rt, n_rt,
       d_real, d_imag);
  }

  void cudatbx::scattering::direct_summation::add
  (const scitbx::af::const_ref<std::string>& scatterers,
   const scitbx::af::const_ref<scitbx::vec3<double> >& xyz,
   const scitbx::af::const_ref<double>& solvent_weights,
   const scitbx::af::const_ref<scitbx::vec3<double> >& h,
   const scitbx::af::const_ref<double>& rotations,
   const scitbx::af::const_ref<scitbx::vec3<double> >& translations,
   const cctbx::xray::scattering_type_registry& registry,
   const bool& complex_form_factor) {

    // reorganize input data, allocates arrays, transfer to GPU, order matters
    reorganize_coordinates(xyz,solvent_weights);
    reorganize_h(h);
    reorganize_rt(rotations,translations);
    convert_scatterers(scatterers,registry,complex_form_factor);

    // allocate arrays for results if necessary
    if (!amplitudes_allocated) {
      allocate_amplitudes();
      reset_amplitudes();
    }

    // run calculation
    run_kernel();

    // deallocate arrays
    clear_arrays();
  }

  /* --------------------------------------------------------------------------
     reorganizes data and calls cuda
     padded to multiple of 128 bytes, (32 * sizeof(float or int))

     "Rapid and accurate calculation of small-angle scattering profiles using
      the golden ratio"
     Watson, MC, Curtis, JE. J. Appl. Cryst. (2013). 46, 1171-1177

     solvent variables are used for weights and code is not optimal
     possibly subclass or split everything into functions
  */
  void cudatbx::scattering::direct_summation::prepare_saxs
  (const scitbx::af::const_ref<std::string>& scatterers,
   const scitbx::af::const_ref<scitbx::vec3<double> >& xyz,
   const scitbx::af::const_ref<double>& solvent_weights,
   const scitbx::af::const_ref<double>& q,
   const scitbx::af::const_ref<double>& lattice_weights,
   const scitbx::af::const_ref<double>& lattice,
   const cctbx::xray::scattering_type_registry& registry,
   const bool& complex_form_factor) {

    // reorganize input data, allocates arrays, transfer to GPU, order matters
    reorganize_coordinates(xyz,solvent_weights);
    reorganize_q(q,lattice_weights,lattice);
    convert_scatterers(scatterers,registry,complex_form_factor);

    // allocate arrays for results if necessary
    if (!amplitudes_allocated) {
      allocate_amplitudes();
    }
  }

  void cudatbx::scattering::direct_summation::run_saxs_kernel() {
    // allocate working space if necessary
    if (!workspace_allocated) {
      workspace_size = int(std::floor(n_h*n_rt/padding + 1.0)) * padding;
      allocate_workspace(3*workspace_size);
    }

    int blocks_per_grid = (n_h*n_rt + threads_per_block - 1)/threads_per_block;
    expand_q_lattice_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (d_h, n_h,
       d_rt, n_rt, size_rt,
       d_workspace, workspace_size);
    saxs_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (d_scattering_type, d_xyz, d_solvent, n_xyz, padded_n_xyz,
       n_h, n_rt,
       d_workspace, workspace_size);
    collect_saxs_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (n_h, n_rt, d_weights,
       d_real, d_imag,
       d_workspace, workspace_size);
  }

  void cudatbx::scattering::direct_summation::run_solvent_saxs_kernel() {
    // allocate working space if necessary
    if (!workspace_allocated) {
      workspace_size = int(std::floor(n_h*n_rt/padding + 1.0)) * padding;
      allocate_workspace(6*workspace_size);
    }

    int blocks_per_grid = (n_h*n_rt + threads_per_block - 1)/threads_per_block;
    expand_q_lattice_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (d_h, n_h,
       d_rt, n_rt, size_rt,
       d_workspace, workspace_size);
    solvent_saxs_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (d_scattering_type, d_xyz, d_solvent, n_xyz, padded_n_xyz,
       n_h, n_rt,
       d_workspace, workspace_size);
  }

  void cudatbx::scattering::direct_summation::run_collect_solvent_saxs_kernel
  (const double& c1, const double& c2) {

    // transfer scaling constants to constant memory on GPU
    fType h_c1 = fType(c1);
    fType h_c2 = fType(c2);
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_c1), &h_c1, sizeof(fType)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_c2), &h_c2, sizeof(fType)) );

    assert(workspace_allocated);
    int blocks_per_grid = (n_h*n_rt + threads_per_block - 1)/threads_per_block;
    collect_solvent_saxs_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (n_h, n_rt, d_weights,
       d_real, d_imag,
       d_workspace, workspace_size);
  }

  /* --------------------------------------------------------------------------
     return total sum
  */
  scitbx::af::shared<std::complex<double> >
  cudatbx::scattering::direct_summation::get_sum() {
    scitbx::af::shared<std::complex<double> > sf(n_h);
    assert(amplitudes_allocated);
    cudaSafeCall( hipMemcpy(h_real,d_real,n_h*sizeof(fType),
                             hipMemcpyDeviceToHost) );
    cudaSafeCall( hipMemcpy(h_imag,d_imag,n_h*sizeof(fType),
                             hipMemcpyDeviceToHost) );
    for (int i=0; i<n_h; i++) {
      sf[i] = std::complex<double>(double(h_real[i]),double(h_imag[i]));
    }

    return sf;
  }

  /* ==========================================================================
   */

}
}
