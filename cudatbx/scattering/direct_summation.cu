#include "hip/hip_runtime.h"
#include <cudatbx/scattering/direct_summation.cuh>

namespace cudatbx {
namespace scattering {

  /* ==========================================================================
   */
  cudatbx::scattering::direct_summation::direct_summation() {
    // set host and device pointers to NULL
    h_xyz = NULL;
    h_solvent = NULL;
    h_h = NULL;
    h_rt = NULL;
    h_scattering_type = NULL;
    h_a = NULL;
    h_b = NULL;
    h_c = NULL;

    d_xyz = NULL;
    d_solvent = NULL;
    d_h = NULL;
    d_rt = NULL;
    d_scattering_type = NULL;
    d_a = NULL;
    d_b = NULL;
    d_c = NULL;

    sf_size = 0;
    sf_real = NULL;
    sf_imag = NULL;
  }

  cudatbx::scattering::direct_summation::~direct_summation() {
    clear_arrays();
    cudaSafeCall( hipFree(sf_real) );
    cudaSafeCall( hipFree(sf_imag) );
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::reorganize_coordinates
  (const scitbx::af::const_ref<scitbx::vec3<double> >& xyz,
   const scitbx::af::const_ref<double>& solvent_weights) {
    n_xyz = xyz.size();
    SCITBX_ASSERT (solvent_weights.size() == n_xyz);
    padded_n_xyz = int(std::floor(n_xyz/padding + 1.0)) * padding;
    size_xyz = 3 * padded_n_xyz;
    delete[] h_xyz;
    delete[] h_solvent;
    h_xyz = new fType[size_xyz];
    h_solvent = new fType[padded_n_xyz];
    for (int i=0; i<n_xyz; i++) {
      for (int j=0; j<3; j++) {
        h_xyz[j*padded_n_xyz + i] = fType(xyz[i][j]);
      }
      h_solvent[i] = solvent_weights[i];
    }

    cudaSafeCall( hipMalloc((void**)&d_xyz,size_xyz*sizeof(fType)) );
    cudaSafeCall( hipMemcpy(d_xyz, h_xyz, size_xyz*sizeof(fType),
                             hipMemcpyHostToDevice) );
    cudaSafeCall( hipMalloc((void**)&d_solvent,padded_n_xyz*sizeof(fType)) );
    cudaSafeCall( hipMemcpy(d_solvent, h_solvent,
                             padded_n_xyz*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::reorganize_h
  (const scitbx::af::const_ref<scitbx::vec3<double> >& h) {
    n_h = h.size();
    padded_n_h = int(std::floor(n_h/padding + 1.0)) * padding;
    size_h = 3 * padded_n_h;
    delete[] h_h;
    h_h = new fType[size_h];
    for (int i=0; i<n_h; i++) {
      for (int j=0; j<3; j++) {
        h_h[j*padded_n_h + i] = fType(h[i][j]);
      }
    }

    cudaSafeCall( hipMalloc((void**)&d_h,size_h*sizeof(fType)) );
    cudaSafeCall( hipMemcpy(d_h, h_h, size_h*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::reorganize_rt
  (const scitbx::af::const_ref<double>& rotations,
   const scitbx::af::const_ref<scitbx::vec3<double> >& translations) {
    // each rotation/translation pair is combined and padded to take up
    // 64 bytes so that a coalesced read will read two pairs
    n_rt = translations.size();
    size_rt = padded_size * n_rt;
    delete[] h_rt;
    h_rt = new fType[size_rt];
    for (int i=0; i<n_rt; i++) {
      for (int j=0; j<9; j++) {
        h_rt[padded_size*i + j] = fType(rotations[9*i + j]);
      }
      for (int j=0; j<3; j++) {
        h_rt[padded_size*i + j + 9] = fType(translations[i][j]);
      }
    }

    cudaSafeCall( hipMalloc((void**)&d_rt,size_rt*sizeof(fType)) );
    cudaSafeCall( hipMemcpy(d_rt, h_rt, size_rt*sizeof(fType),
                             hipMemcpyHostToDevice) );
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::convert_scatterers
  (const scitbx::af::const_ref<std::string>& scatterers,
   const cctbx::xray::scattering_type_registry& registry,
   const bool& complex_form_factor) {
    // convert scattering types and form factors
    // add ordinary oxygen form factor at end for boundary layer solvent
    delete[] h_scattering_type;
    h_scattering_type = new int[padded_n_xyz];
    for (int i=0; i<n_xyz; i++) {
      h_scattering_type[i] = registry.unique_index(scatterers[i]);
    }
    scitbx::af::shared<boost::optional
                       <cctbx::eltbx::xray_scattering::gaussian> >
      unique_gaussians = registry.unique_gaussians;
    n_types = unique_gaussians.size() + 1;
    n_terms = unique_gaussians[0].get().n_terms();
    f_size = n_types * n_terms;
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    h_a = new fType[f_size];
    h_b = new fType[f_size];
    h_c = new fType[n_types];
    for (int i=0; i<f_size; i++) {
      h_a[i] = fType(0.0);
      h_b[i] = fType(0.0);
    }
    for (int i=0; i<n_types-1; i++) {
      for (int j=0; j<n_terms; j++) {
        h_a[i*n_terms + j] = unique_gaussians[i].get().array_of_a()[j];
        h_b[i*n_terms + j] = unique_gaussians[i].get().array_of_b()[j];
      }
      if (unique_gaussians[i].get().use_c()) {
        h_c[i] = unique_gaussians[i].get().c();
      }
      else {
        h_c[i] = fType(0.0);
      }
    }

    // add form factor for boundary layer solvent
    cctbx::eltbx::xray_scattering::gaussian hoh =
      unique_gaussians[registry.unique_index("O")].get();
    for (int i=0; i<hoh.array_of_a().size(); i++){
      h_a[(n_types-1)*n_terms + i] = hoh.array_of_a()[i];
      h_b[(n_types-1)*n_terms + i] = hoh.array_of_b()[i];
    }
    if (hoh.use_c()) {
      h_c[n_types-1] = hoh.c();
    }
    else {
      h_c[n_types-1] = fType(0.0);
    }

    cudaSafeCall( hipMalloc((void**)&d_scattering_type,padded_n_xyz*sizeof(int)) );
    cudaSafeCall( hipMemcpy(d_scattering_type,h_scattering_type,
                             padded_n_xyz*sizeof(int),hipMemcpyHostToDevice) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_a), h_a, f_size*sizeof(fType)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_b), h_b, f_size*sizeof(fType)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_c), h_c, n_types*sizeof(fType)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_n_types), &n_types, sizeof(int)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_n_terms), &n_terms, sizeof(int)) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dc_complex_form_factor),
                                     &complex_form_factor, sizeof(bool)) );
  }

  // --------------------------------------------------------------------------
  void cudatbx::scattering::direct_summation::clear_arrays() {
    // all pointers are NULL or point to allocated arrays
    delete[] h_xyz;
    delete[] h_solvent;
    delete[] h_h;
    delete[] h_rt;
    delete[] h_scattering_type;
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    cudaSafeCall( hipFree(d_xyz) );
    cudaSafeCall( hipFree(d_solvent) );
    cudaSafeCall( hipFree(d_h) );
    cudaSafeCall( hipFree(d_rt) );
    cudaSafeCall( hipFree(d_scattering_type) );
    cudaSafeCall( hipFree(d_a) );
    cudaSafeCall( hipFree(d_b) );
    cudaSafeCall( hipFree(d_c) );

    // reset all pointers to NULL
    h_xyz = NULL;
    h_solvent = NULL;
    h_h = NULL;
    h_rt = NULL;
    h_scattering_type = NULL;
    h_a = NULL;
    h_b = NULL;
    h_c = NULL;

    d_xyz = NULL;
    d_solvent = NULL;
    d_h = NULL;
    d_rt = NULL;
    d_scattering_type = NULL;
    d_a = NULL;
    d_b = NULL;
    d_c = NULL;
  }

  /* --------------------------------------------------------------------------
     reorganizes data and calls cuda
     padded to multiple of 128 bytes, (32 * sizeof(float or int))
  */
  void cudatbx::scattering::direct_summation::add
  (const scitbx::af::const_ref<std::string>& scatterers,
   const scitbx::af::const_ref<scitbx::vec3<double> >& xyz,
   const scitbx::af::const_ref<double>& solvent_weights,
   const scitbx::af::const_ref<scitbx::vec3<double> >& h,
   const scitbx::af::const_ref<double>& rotations,
   const scitbx::af::const_ref<scitbx::vec3<double> >& translations,
   const cctbx::xray::scattering_type_registry& registry,
   const bool& complex_form_factor) {

    // reorganize input data, allocates arrays, transfer to GPU, order matters
    reorganize_coordinates(xyz,solvent_weights);
    reorganize_h(h);
    reorganize_rt(rotations,translations);
    convert_scatterers(scatterers,registry,complex_form_factor);

    // allocate arrays for results if necessary
    if (sf_size == 0) {
      sf_size = n_h;
      cudaSafeCall( hipMalloc((void**)&sf_real,n_h*sizeof(fType)) );
      cudaSafeCall( hipMalloc((void**)&sf_imag,n_h*sizeof(fType)) );
    }
    else {
      assert(sf_size == n_h);
    }

    // run calculation
    int blocks_per_grid = (n_h + threads_per_block - 1)/threads_per_block;
    structure_factor_kernel<fType><<<blocks_per_grid,threads_per_block>>>
      (d_scattering_type, d_xyz, d_solvent, n_xyz, padded_n_xyz,
       d_h, n_h, padded_n_h,
       d_rt, n_rt,
       sf_real, sf_imag);

    // deallocate arrays
    clear_arrays();
  }

  /* --------------------------------------------------------------------------
     return total sum
  */
  scitbx::af::shared<std::complex<double> >
  cudatbx::scattering::direct_summation::get_sum() {
    scitbx::af::shared<std::complex<double> > sf(sf_size);
    if (sf_size != 0) {
      fType* h_real = new fType[sf_size];
      fType* h_imag = new fType[sf_size];
      cudaSafeCall( hipMemcpy(h_real,sf_real,sf_size*sizeof(fType),
                               hipMemcpyDeviceToHost) );
      cudaSafeCall( hipMemcpy(h_imag,sf_imag,sf_size*sizeof(fType),
                               hipMemcpyDeviceToHost) );
      for (int i=0; i<sf_size; i++) {
        sf[i] = std::complex<double>(double(h_real[i]),double(h_imag[i]));
      }
      delete[] h_real;
      delete[] h_imag;
    }
    return sf;
  }

  /* ==========================================================================
   */

}
}
